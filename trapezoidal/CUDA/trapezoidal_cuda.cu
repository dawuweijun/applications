#include "hip/hip_runtime.h"
/* Matheus da Silva Serpa - Ciência da Computação (2015)
 * Universidade Federal do Pampa - Campus Alegrete
 * matheusserpa@gmail.com
 * https://github.com/matheusserpa/applications */
 
#include <stdio.h>
#include <stdlib.h>
#include "../../boltzmann/CUDA/helper.cuh"
#include <hip/hip_runtime.h>

static void GPUHandleError( hipError_t err, const char *file, const int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}

#define GPU_HANDLE_ERROR( err ) (GPUHandleError( err, __FILE__, __LINE__ ))

	long long int n;
	int threadsPerBlock, blocksPerGrid;

#ifdef USE_DOUBLE
 	__host__ __device__ double f(double x)
#else
 	__host__ __device__ float f(float x)
#endif
 	{
 		return x;
 	}


#ifdef USE_DOUBLE
	__global__ void sum_reduct(double a, double h, double *sum, long long int n)
#else
	__global__ void sum_reduct(float a, float h, float *sum, long long int n)
#endif
	{
		#ifdef USE_DOUBLE
			__shared__ double cache[256];
			double temp = 0.0;
		#else
			__shared__ float cache[256];
			float temp = 0.0;
		#endif

		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		int cacheIdx = threadIdx.x;

		while(tid < n){
			temp += f(a + tid * h);
			tid += blockDim.x * gridDim.x;
		}

		// set the cache values
		cache[cacheIdx] = temp;

		// synchronize threads in this block
		__syncthreads();

		//for reductions, threadsPerBlock must be a power of 2
		int i = blockDim.x / 2;
		while(i != 0 ){
			if(cacheIdx < i)
				cache[cacheIdx] += cache[cacheIdx + i];
			__syncthreads();
			i /= 2;
		}

		if(cacheIdx == 0)
			sum[blockIdx.x] = cache[0];

	}

int main(int argc, char **argv){

	if(checkFlag(argv, argc, "debug")){
		#ifdef USE_DOUBLE
			printf("Trapezoidal Rule - CUDA - Double Precision\n");
		#else
			printf("Trapezoidal Rule - CUDA - Single Precision\n");
		#endif
		printf("Serpa and Schepke 2015\n");
		printf("Laboratório de Estudos Avançados - UNIPAMPA\n\n");
	}


	#ifdef USE_DOUBLE
		double a, b, I, h;
	#else		 
		float a, b, I, h;
	#endif

	if(flagValueLong(argv, argc, "n") > 0)
		n = flagValueLong(argv, argc, "n");
	else
		n = 10;

	if(flagValueReal(argv, argc, "a") > 0){
		a = flagValueReal(argv, argc, "a");
	}
	else
		a = 1;

	if(flagValueReal(argv, argc, "b") > 0)
		b = flagValueReal(argv, argc, "b");
	else
		b = 2;

	h = (b - a) / n;

	//Execution time
	float timer = 0.0;
    
    if(checkFlag(argv, argc, "debug")){
    	printf("Begin main loop\n");
    }

   	// Workstation UNIPAMPA: 0 to Tesla C2075, 1 to Quadro 5000
	GPU_HANDLE_ERROR(hipSetDevice(0));

	// Device memory and Host memory
	#ifdef USE_DOUBLE
		double *h_sum, *d_sum;
	#else
		float  *h_sum, *d_sum;
	#endif

	// Blocks and Grids
	threadsPerBlock = 256; // remember to chance in cuda kernel
	blocksPerGrid = 32;

	if(((n - 2 + threadsPerBlock - 1) / threadsPerBlock) < 32)
		blocksPerGrid = (n - 2 + threadsPerBlock - 1) / threadsPerBlock;

	if(checkFlag(argv, argc, "debug")){
		printf("\n");
		#ifdef USE_DOUBLE
			printf("[%.2lf %.2lf] ---- n = %lld --- h = %.10lf\n", a, b, n, h);
		#else
			printf("[%.2f %.2f] ---- n = %lld --- h = %.10f\n", a, b, n, h);
		#endif
		printf("threadsPerBlock = %d --- blocksPerGrid = %d\n\n", threadsPerBlock, blocksPerGrid);
	}



	// Timer
	hipEvent_t start, stop;

	GPU_HANDLE_ERROR(hipEventCreate(&start));
	GPU_HANDLE_ERROR(hipEventCreate(&stop));

	// Memory alloc for host and device
	#ifdef USE_DOUBLE	
		GPU_HANDLE_ERROR(hipMalloc((void **) &d_sum,  blocksPerGrid * sizeof(double)));
		h_sum = (double *) malloc(blocksPerGrid * sizeof(double));
	#else
		GPU_HANDLE_ERROR(hipMalloc((void **) &d_sum,  blocksPerGrid * sizeof(float)));
		h_sum = (double *) malloc(blocksPerGrid * sizeof(float));
	#endif

    I = 0;

    // CUDA Code
	GPU_HANDLE_ERROR(hipDeviceSynchronize());

	GPU_HANDLE_ERROR(hipEventRecord(start, 0));
		sum_reduct<<<blocksPerGrid, threadsPerBlock>>>(a, h, d_sum, n - 2);
	GPU_HANDLE_ERROR(hipEventRecord(stop, 0));	GPU_HANDLE_ERROR(hipEventSynchronize(stop));

  	GPU_HANDLE_ERROR(hipDeviceSynchronize()); GPU_HANDLE_ERROR(hipGetLastError());

    GPU_HANDLE_ERROR(hipEventElapsedTime(&timer, start, stop));
    
    // Converting miliseconds to seconds
	timer /= 1000;

	// Stop events
  	GPU_HANDLE_ERROR(hipEventDestroy(start));
	GPU_HANDLE_ERROR(hipEventDestroy(stop));

	// Copy device to host
	#ifdef USE_DOUBLE
		GPU_HANDLE_ERROR(hipMemcpy(h_sum, d_sum, blocksPerGrid * sizeof(double), hipMemcpyDeviceToHost));
	#else
		GPU_HANDLE_ERROR(hipMemcpy(h_sum, d_sum, blocksPerGrid * sizeof(float),  hipMemcpyDeviceToHost));
	#endif

	// Synchronize
	GPU_HANDLE_ERROR(hipDeviceSynchronize());

	// finish up on the CPU side
	int i;
	I = 0;
	for(i = 0; i < blocksPerGrid; i++)
		I += h_sum[i];

	// CUDA Code again
	// Free device memory
	GPU_HANDLE_ERROR(hipFree(d_sum));

	// Free host memory
	free(h_sum);

	// Device Reset
	GPU_HANDLE_ERROR(hipDeviceReset());

	if(checkFlag(argv, argc, "debug"))
		#ifdef USE_DOUBLE
			printf("GPU_sum = %.10lf\n", I);
		#else
			printf("GPU_sum = %.10f\n", I);
		#endif

	I = (h / 2) * (f(a) + 2 * I + f(b));

	if(checkFlag(argv, argc, "debug")){
	    printf("Finish main loop\n");

	    #ifdef USE_DOUBLE
	    	printf("\nTrapezoidal = %.10lf\n", I);
	    #else
	    	printf("\nTrapezoidal = %.10f\n", I);
	    #endif

	    printf("Time\n");

	    printf("\texecution time: %.10lf segs.\n", timer);

	    printf("End of the execution\n\n");
	}	

	if(checkFlag(argv, argc, "check") && a == 1 && b == 2){
		#ifdef USE_DOUBLE
	    	printf("%.10lf\n", I);
	    #else
	    	printf("%.10f\n", I);
	    #endif
		printf("OK\n1.5\n\n");
	}

    fprintf(stderr, "%.10lf\n", timer);

	return 0;
}