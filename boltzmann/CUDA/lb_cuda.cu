#include "hip/hip_runtime.h"
/* Matheus da Silva Serpa - Ciência da Computação (2015)
 * Universidade Federal do Pampa - Campus Alegrete
 * matheusserpa@gmail.com
 * https://github.com/matheusserpa/applications */

#include <stdio.h>
#include <stdlib.h>
#include "helper.cuh"
#include "lb_3D.cuh"
 #include <hip/hip_runtime.h>

static void GPUHandleError( hipError_t err, const char *file, const int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}

#define GPU_HANDLE_ERROR( err ) (GPUHandleError( err, __FILE__, __LINE__ ))

/* It is interesting to redistribute de forces to all points */
#ifdef USE_DOUBLE
	__global__ void redistribute(unsigned short int *obst, double *node, unsigned short int lx, unsigned short int ly, unsigned short int lz)
#else
	__global__ void redistribute(unsigned short int *obst, float *node, unsigned short int lx, unsigned short int ly, unsigned short int lz)
#endif
	{
		unsigned short int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned short int y = blockIdx.y * blockDim.y + threadIdx.y;
		unsigned short int z = blockIdx.z * blockDim.z + threadIdx.z;
		
		if(x < lx && y < ly && z < lz && obst[x * ly * lz + y * lz + z] == false){				
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 1]  += T_1;
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 2]  += T_2;
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 4]  -= T_2;
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 5]  -= T_1;
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 6]  -= T_2;
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 8]  += T_2;
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 9]  += T_2;
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 11] -= T_2;
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 12] -= T_2;
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 14] += T_2;
		}
	}


//////////////////////////////////////////
// Propagate
//////////////////////////////////////////
#ifdef USE_DOUBLE
	__global__ void propagate(double *node, double *temp, unsigned short int lx, unsigned short int ly, unsigned short int lz)
#else
	__global__ void propagate(float *node, float *temp, unsigned short int lx, unsigned short int ly, unsigned short int lz)
#endif
	{
		unsigned short int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned short int y = blockIdx.y * blockDim.y + threadIdx.y;
		unsigned short int z = blockIdx.z * blockDim.z + threadIdx.z;
				
		if(x < lx && y < ly && z < lz){
			unsigned short int x_e, x_w, y_u, y_d, z_n, z_s;

			//compute upper and right next neighbour nodes
			x_e = (x + 1) % lx;
			y_u = (y + 1) % ly;
			z_n = (z + 1) % lz;

			//compute lower and left next neighbour nodes
			x_w = (x - 1 + lx) % lx;
			y_d = (y - 1 + ly) % ly;
			z_s = (z - 1 + lz) % lz;
			//density propagation
			
			//zero
			temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 0]  = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 0];

			//east
			temp[x_e * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 1]  = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 1];
			//north
			temp[x_e * ly * lz * NDIM + y_u * lz * NDIM + z * NDIM + 2]  = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 2];
			//west
			temp[x * ly * lz * NDIM + y_u * lz * NDIM + z * NDIM + 3]  = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 3];
			//south
			temp[x_w * ly * lz * NDIM + y_u * lz * NDIM + z * NDIM + 4]  = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 4];
			//down
			temp[x_w * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 5]  = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 5];
			//up
			temp[x_w * ly * lz * NDIM + y_d * lz * NDIM + z * NDIM + 6]  = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 6];

			//east down
			temp[x * ly * lz * NDIM + y_d * lz * NDIM + z * NDIM + 7]  = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 7];
			//east up
			temp[x_e * ly * lz * NDIM + y_d * lz * NDIM + z * NDIM + 8]  = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 8];
			//north-east
			temp[x_e * ly * lz * NDIM + y * lz * NDIM + z_n * NDIM + 9]  = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 9];
			//north-down
			temp[x * ly * lz * NDIM + y * lz * NDIM + z_n * NDIM + 10] = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 10];
			//north-up
			temp[x_w * ly * lz * NDIM + y * lz * NDIM + z_n * NDIM + 11] = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 11];
			//north-west
			temp[x_w * ly * lz * NDIM + y * lz * NDIM + z_s * NDIM + 12] = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 12];
			//west down
			temp[x * ly * lz * NDIM + y * lz * NDIM + z_s * NDIM + 13] = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 13];
			//west up
			temp[x_e * ly * lz * NDIM + y * lz * NDIM + z_s * NDIM + 14] = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 14];
			//south-west
			temp[x * ly * lz * NDIM + y_u * lz * NDIM + z_n * NDIM + 15] = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 15];
			//south down
			temp[x * ly * lz * NDIM + y_d * lz * NDIM + z_n * NDIM + 16] = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 16];
			//south up
			temp[x * ly * lz * NDIM + y_d * lz * NDIM + z_s * NDIM + 17] = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 17];
			//south-east
			temp[x * ly * lz * NDIM + y_u * lz * NDIM + z_s * NDIM + 18] = node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 18];
		}
	}

//////////////////////////////////////////
// Bounceback
//////////////////////////////////////////
#ifdef USE_DOUBLE
	__global__ void bounceback(unsigned short int *obst, double *node, double *temp, unsigned short int lx, unsigned short int ly, unsigned short int lz)
#else
	__global__ void bounceback(unsigned short int *obst, float *node, double *temp, unsigned short int lx, unsigned short int ly, unsigned short int lz)
#endif
	{
		unsigned short int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned short int y = blockIdx.y * blockDim.y + threadIdx.y;
		unsigned short int z = blockIdx.z * blockDim.z + threadIdx.z;
				
		if(x < lx && y < ly && z < lz && obst[x * ly * lz + y * lz + z] == true){
			//east
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 1]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 5];
			//north
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 2]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 6];
			//west
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 3]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 7];
			//south
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 4]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 8];
			//down
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 5]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 1];
			//up
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 6]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 2];
			
			//east down
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 7]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 3];
			//east up
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 8]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 4];
			//north-east
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 9]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 12];
			//north-down
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 10] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 13];
			//north-up
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 11] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 14];
			//north-west
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 12] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 9];
			//west down
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 13] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 10];
			//west up
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 14] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 11];
			//south-west
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 15] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 17];
			//south down
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 16] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 18];
			//south up
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 17] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 15];
			//south-east
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 18] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 16];
		}
	}


//////////////////////////////////////////
// Relaxation
//////////////////////////////////////////
#ifdef USE_DOUBLE
	__global__ void relaxation(unsigned short int *obst, double *node, double *temp, unsigned short int lx, unsigned short int ly, unsigned short int lz)
#else
	__global__ void relaxation(unsigned short int *obst, float *node, double *temp, unsigned short int lx, unsigned short int ly, unsigned short int lz)
#endif
	{
		unsigned short int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned short int y = blockIdx.y * blockDim.y + threadIdx.y;
		unsigned short int z = blockIdx.z * blockDim.z + threadIdx.z;
				
		if(x < lx && y < ly && z < lz && obst[x * ly * lz + y * lz + z] == false){
			#ifdef USE_DOUBLE
				double u_x, u_y, u_z;
				double u_n[NDIM], n_equ[NDIM], u_squ, d_loc;
			#else
				float u_x, u_y, u_z;
				float u_n[NDIM], n_equ[NDIM], u_squ, d_loc;
			#endif

			d_loc = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 0]  + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 1]  + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 2]  + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 3]  + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 4]  + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 5]
				  + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 6]  + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 7]  + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 8]  + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 9]  + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 10] + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 11]
				  + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 12] + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 13] + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 14] + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 15] + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 16] + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 17] 
				  + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 18];

			//x-, y- and z- velocity components
			u_x = (temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 1] + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 2] + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 8] + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 9]  + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 14]
			      -temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 4] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 5] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 6] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 11] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 13]
				  ) / d_loc;

			u_y = (temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 2] + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 3] + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 4] + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 15] + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 18]
				  -temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 5] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 7] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 8] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 16] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 17]
				  ) / d_loc;
			
			u_z = (temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 9]  + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 10] + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 11] + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 15] + temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 16]
				  -temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 12] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 13] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 14] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 17] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 18]
				  ) / d_loc;

			//square velocity
			u_squ = u_x * u_x + u_y * u_y + u_z * u_z;

			//n- velocity components
			//only 3 speeds would be necessary
			u_n[1]  =  u_x;
			u_n[2]  =  u_x + u_y;
			u_n[3]  =  u_y;
			u_n[4]  = -u_x + u_y;
			u_n[5]  = -u_x;
			u_n[6]  = -u_x - u_y;
			u_n[7]  = -u_y;
			u_n[8]  =  u_x - u_y;

			u_n[9]  =  u_x + u_z;
			u_n[10] =  u_z;
			u_n[11] = -u_x + u_z;
			u_n[12] = -u_x - u_z;
			u_n[13] = -u_z;
			u_n[14] =  u_x - u_z;
			u_n[15] =  u_y + u_z;
			u_n[16] = -u_y + u_z;
			u_n[17] = -u_y - u_z;
			u_n[18] =  u_y - u_z;
					  
			//zero velocity density
			n_equ[0]  = T0 * d_loc * (1.0 - u_squ   / (2.0 * CS2));

			//axis speeds: factor: t1
			n_equ[1]  = T1 * d_loc * (1.0 + u_n[1]  / CS2 + u_n[1]  * u_n[1]  / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			n_equ[2]  = T2 * d_loc * (1.0 + u_n[2]  / CS2 + u_n[2]  * u_n[2]  / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			n_equ[3]  = T1 * d_loc * (1.0 + u_n[3]  / CS2 + u_n[3]  * u_n[3]  / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			n_equ[4]  = T2 * d_loc * (1.0 + u_n[4]  / CS2 + u_n[4]  * u_n[4]  / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			n_equ[5]  = T1 * d_loc * (1.0 + u_n[5]  / CS2 + u_n[5]  * u_n[5]  / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			n_equ[6]  = T2 * d_loc * (1.0 + u_n[6]  / CS2 + u_n[6]  * u_n[6]  / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			
			//diagonal speeds: factor t2
			n_equ[7]  = T1 * d_loc * (1.0 + u_n[7]  / CS2 + u_n[7]  * u_n[7]  / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			n_equ[8]  = T2 * d_loc * (1.0 + u_n[8]  / CS2 + u_n[8]  * u_n[8]  / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			n_equ[9]  = T2 * d_loc * (1.0 + u_n[9]  / CS2 + u_n[9]  * u_n[9]  / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			n_equ[10] = T1 * d_loc * (1.0 + u_n[10] / CS2 + u_n[10] * u_n[10] / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			n_equ[11] = T2 * d_loc * (1.0 + u_n[11] / CS2 + u_n[11] * u_n[11] / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			n_equ[12] = T2 * d_loc * (1.0 + u_n[12] / CS2 + u_n[12] * u_n[12] / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			n_equ[13] = T1 * d_loc * (1.0 + u_n[13] / CS2 + u_n[13] * u_n[13] / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			n_equ[14] = T2 * d_loc * (1.0 + u_n[14] / CS2 + u_n[14] * u_n[14] / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			n_equ[15] = T2 * d_loc * (1.0 + u_n[15] / CS2 + u_n[15] * u_n[15] / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			n_equ[16] = T2 * d_loc * (1.0 + u_n[16] / CS2 + u_n[16] * u_n[16] / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			n_equ[17] = T2 * d_loc * (1.0 + u_n[17] / CS2 + u_n[17] * u_n[17] / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
			n_equ[18] = T2 * d_loc * (1.0 + u_n[18] / CS2 + u_n[18] * u_n[18] / (2.0 * CS2 * CS2) - u_squ / (2.0 * CS2));
		
			// relaxation step
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 0]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 0]  + OMEGA * (n_equ[0]  - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 0] );
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 1]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 1]  + OMEGA * (n_equ[1]  - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 1] );
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 2]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 2]  + OMEGA * (n_equ[2]  - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 2] );
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 3]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 3]  + OMEGA * (n_equ[3]  - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 3] );
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 4]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 4]  + OMEGA * (n_equ[4]  - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 4] );
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 5]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 5]  + OMEGA * (n_equ[5]  - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 5] );
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 6]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 6]  + OMEGA * (n_equ[6]  - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 6] );
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 7]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 7]  + OMEGA * (n_equ[7]  - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 7] );
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 8]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 8]  + OMEGA * (n_equ[8]  - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 8] );
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 9]  = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 9]  + OMEGA * (n_equ[9]  - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 9] );
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 10] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 10] + OMEGA * (n_equ[10] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 10]);
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 11] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 11] + OMEGA * (n_equ[11] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 11]);
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 12] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 12] + OMEGA * (n_equ[12] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 12]);
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 13] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 13] + OMEGA * (n_equ[13] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 13]);
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 14] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 14] + OMEGA * (n_equ[14] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 14]);
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 15] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 15] + OMEGA * (n_equ[15] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 15]);
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 16] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 16] + OMEGA * (n_equ[16] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 16]);
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 17] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 17] + OMEGA * (n_equ[17] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 17]);
			node[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 18] = temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 18] + OMEGA * (n_equ[18] - temp[x * ly * lz * NDIM + y * lz * NDIM + z * NDIM + 18]);	
		}
	}

int main(int argc, char **argv){

	int block;
	if(flagValueInt(argv, argc, "block") > 0){
		block = flagValueInt(argv, argc, "block");
	}
	else{
		block = 8;
	}	
	
	if(checkFlag(argv, argc, "debug")){
		#ifdef USE_DOUBLE
			printf("Lattice Boltzmann Method - D3Q19 - Vector - CUDA - Block: %d - Double Precision\n", block);
		#else
			printf("Lattice Boltzmann Method - D3Q19 - Vector - CUDA - Block: %d - Single Precision\n", block);
		#endif
		printf("Serpa and Schepke 2015\n");
		printf("Laboratório de Estudos Avançados - UNIPAMPA\n\n");
	}

	/* Iteration counter */
	unsigned short int time;

	//Execution time
	float timer[6] = {0.0};

	//Check error
	double e_i = 0.0, e_j = 0.0; /* Iteration i and j error */
	double e_r = 0.0; /* Relative error */
	double const e_s = 0.5E-10; /* Estimated error */

	/* Properties structure */
	print_parameters(argv, argc);

	/* Lattice structure */
	s_lattice *lattice =(s_lattice*) read_obstacles(argv, argc);

	
	init_density(lattice, checkFlag(argv, argc, "debug"));

	if(checkFlag(argv, argc, "check")){
		e_i = check_density(lattice, checkFlag(argv, argc, "debug"));
	}
	
	if(checkFlag(argv, argc, "debug")){
		printf("Start main loop\n");
	}

	// Workstation UNIPAMPA: 0 to Tesla C2075, 1 to Quadro 5000
	//GPU_HANDLE_ERROR(hipSetDevice(0));

	// Device Memory
	unsigned short int *d_obst;
	#ifdef USE_DOUBLE
		double *d_node, *d_temp;
	#else
		float *d_node, *d_temp;
	#endif

	// Blocks and Grids.
	dim3 BLOCK_REDISTRIBUTE(block, block, block);
	dim3 BLOCK_PROPAGATE(block, block, block);
	dim3 BLOCK_BOUNCEBACK(block, block, block);
	dim3 BLOCK_RELAXATION(block, block, block);

	int a = (lattice->lx + BLOCK_REDISTRIBUTE.x - 1) / BLOCK_REDISTRIBUTE.x;
	int b = (lattice->ly + BLOCK_REDISTRIBUTE.y - 1) / BLOCK_REDISTRIBUTE.y;
	int c = (lattice->lz + BLOCK_REDISTRIBUTE.z - 1) / BLOCK_REDISTRIBUTE.z;
	dim3 GRID_REDISTRIBUTE(a, b, c);

	a = (lattice->lx + BLOCK_PROPAGATE.x - 1) / BLOCK_PROPAGATE.x;
	b = (lattice->ly + BLOCK_PROPAGATE.y - 1) / BLOCK_PROPAGATE.y;
	c = (lattice->lz + BLOCK_PROPAGATE.z - 1) / BLOCK_PROPAGATE.z;
	dim3 GRID_PROPAGATE(a, b, c);

	a = (lattice->lx + BLOCK_BOUNCEBACK.x - 1) / BLOCK_BOUNCEBACK.x;
	b = (lattice->ly + BLOCK_BOUNCEBACK.y - 1) / BLOCK_BOUNCEBACK.y;
	c = (lattice->lz + BLOCK_BOUNCEBACK.z - 1) / BLOCK_BOUNCEBACK.z;
	dim3 GRID_BOUNCEBACK(a, b, c);

	a = (lattice->lx + BLOCK_RELAXATION.x - 1) / BLOCK_RELAXATION.x;
	b = (lattice->ly + BLOCK_RELAXATION.y - 1) / BLOCK_RELAXATION.y;
	c = (lattice->lz + BLOCK_RELAXATION.z - 1) / BLOCK_RELAXATION.z;
	dim3 GRID_RELAXATION(a, b, c);
	
	// Timer
	hipEvent_t start[2], stop[2];

	GPU_HANDLE_ERROR(hipEventCreate(&start[0]));
	GPU_HANDLE_ERROR(hipEventCreate(&start[1]));

	GPU_HANDLE_ERROR(hipEventCreate(&stop[0]));
	GPU_HANDLE_ERROR(hipEventCreate(&stop[1]));

	// Memory alloc
	GPU_HANDLE_ERROR(hipMalloc((void **) &d_obst, lattice->lx * lattice->ly * lattice->lz * sizeof(unsigned short int)));
	#ifdef USE_DOUBLE
		GPU_HANDLE_ERROR(hipMalloc((void **) &d_node, lattice->lx * lattice->ly * lattice->lz * NDIM * sizeof(double)));
		GPU_HANDLE_ERROR(hipMalloc((void **) &d_temp, lattice->lx * lattice->ly * lattice->lz * NDIM * sizeof(double)));
	#else
		GPU_HANDLE_ERROR(hipMalloc((void **) &d_node, lattice->lx * lattice->ly * lattice->lz * NDIM * sizeof(float)));
		GPU_HANDLE_ERROR(hipMalloc((void **) &d_temp, lattice->lx * lattice->ly * lattice->lz * NDIM * sizeof(float)));
	#endif

	// Memory Copy
	GPU_HANDLE_ERROR(hipMemcpy(d_obst, lattice->obst, lattice->lx * lattice->ly * lattice->lz * sizeof(unsigned short int), hipMemcpyHostToDevice));
	#ifdef USE_DOUBLE
		GPU_HANDLE_ERROR(hipMemcpy(d_node, lattice->node, lattice->lx * lattice->ly * lattice->lz * NDIM * sizeof(double), hipMemcpyHostToDevice));
		GPU_HANDLE_ERROR(hipMemcpy(d_temp, lattice->temp, lattice->lx * lattice->ly * lattice->lz * NDIM * sizeof(double), hipMemcpyHostToDevice));
	#else
		GPU_HANDLE_ERROR(hipMemcpy(d_node, lattice->node, lattice->lx * lattice->ly * lattice->lz * NDIM * sizeof(float), hipMemcpyHostToDevice));
		GPU_HANDLE_ERROR(hipMemcpy(d_temp, lattice->temp, lattice->lx * lattice->ly * lattice->lz * NDIM * sizeof(float), hipMemcpyHostToDevice));
	#endif

	// Synchronize
	GPU_HANDLE_ERROR(hipDeviceSynchronize());

	GPU_HANDLE_ERROR(hipEventRecord(start[0], 0));
	for(time = 1; time <= ITERATIONS; time++){
		GPU_HANDLE_ERROR(hipEventRecord(start[1], 0));
	    	redistribute<<<GRID_REDISTRIBUTE, BLOCK_REDISTRIBUTE>>>(d_obst, d_node, lattice->lx, lattice->ly, lattice->lz);
	    GPU_HANDLE_ERROR(hipDeviceSynchronize()); GPU_HANDLE_ERROR(hipGetLastError());
	    GPU_HANDLE_ERROR(hipEventRecord(stop[1], 0));	GPU_HANDLE_ERROR(hipEventSynchronize(stop[1]));
  		GPU_HANDLE_ERROR(hipEventElapsedTime(&timer[AUX], start[1], stop[1]));
	    timer[REDISTRIBUTE] += timer[AUX];

	    GPU_HANDLE_ERROR(hipEventRecord(start[1], 0));
	    	propagate<<<GRID_PROPAGATE, BLOCK_PROPAGATE>>>(d_node, d_temp, lattice->lx, lattice->ly, lattice->lz);
	    GPU_HANDLE_ERROR(hipDeviceSynchronize()); GPU_HANDLE_ERROR(hipGetLastError());
	    GPU_HANDLE_ERROR(hipEventRecord(stop[1], 0));	GPU_HANDLE_ERROR(hipEventSynchronize(stop[1]));
  		GPU_HANDLE_ERROR(hipEventElapsedTime(&timer[AUX], start[1], stop[1]));
	    timer[PROPAGATE] += timer[AUX];

	    GPU_HANDLE_ERROR(hipEventRecord(start[1], 0));
	    	bounceback<<<GRID_BOUNCEBACK, BLOCK_BOUNCEBACK>>>(d_obst, d_node, d_temp, lattice->lx, lattice->ly, lattice->lz);
	    GPU_HANDLE_ERROR(hipDeviceSynchronize()); GPU_HANDLE_ERROR(hipGetLastError());
	    GPU_HANDLE_ERROR(hipEventRecord(stop[1], 0));	GPU_HANDLE_ERROR(hipEventSynchronize(stop[1]));
  		GPU_HANDLE_ERROR(hipEventElapsedTime(&timer[AUX], start[1], stop[1]));
	    timer[BOUNCEBACK] += timer[AUX];

	    GPU_HANDLE_ERROR(hipEventRecord(start[1], 0));
	    	relaxation<<<GRID_RELAXATION, BLOCK_RELAXATION>>>(d_obst, d_node, d_temp, lattice->lx, lattice->ly, lattice->lz);
	    GPU_HANDLE_ERROR(hipDeviceSynchronize()); GPU_HANDLE_ERROR(hipGetLastError());
	    GPU_HANDLE_ERROR(hipEventRecord(stop[1], 0));	GPU_HANDLE_ERROR(hipEventSynchronize(stop[1]));
  		GPU_HANDLE_ERROR(hipEventElapsedTime(&timer[AUX], start[1], stop[1]));
	    timer[RELAXATION] += timer[AUX];

	    if(checkFlag(argv, argc, "check")){
	    	printf("%d - ", time);
			check_density(lattice, checkFlag(argv, argc, "debug"));
		}
	}
	GPU_HANDLE_ERROR(hipEventRecord(stop[0], 0));	GPU_HANDLE_ERROR(hipEventSynchronize(stop[0]));
  	GPU_HANDLE_ERROR(hipDeviceSynchronize()); GPU_HANDLE_ERROR(hipGetLastError());
    GPU_HANDLE_ERROR(hipEventElapsedTime(&timer[EXECUTION], start[0], stop[0]));

	// Converting miliseconds to seconds
	for(unsigned int i = 0; i < 5; ++i)
		timer[i] /= 1000;

  	// Stop events
  	GPU_HANDLE_ERROR(hipEventDestroy(start[0]));
	GPU_HANDLE_ERROR(hipEventDestroy(start[1]));

	GPU_HANDLE_ERROR(hipEventDestroy(stop[0]));
	GPU_HANDLE_ERROR(hipEventDestroy(stop[1]));

	// Copy device to host
	#ifdef USE_DOUBLE
	GPU_HANDLE_ERROR(hipMemcpy(lattice->node, d_node, lattice->lx * lattice->ly * lattice->lz * NDIM * sizeof(double), hipMemcpyDeviceToHost));
	#else
	GPU_HANDLE_ERROR(hipMemcpy(lattice->node, d_node, lattice->lx * lattice->ly * lattice->lz * NDIM * sizeof(float), hipMemcpyDeviceToHost));
	#endif

	// Synchronize
	GPU_HANDLE_ERROR(hipDeviceSynchronize());

	// Free device memory
	GPU_HANDLE_ERROR(hipFree(d_obst));
	GPU_HANDLE_ERROR(hipFree(d_node));
	GPU_HANDLE_ERROR(hipFree(d_temp));

	// Device Reset
	GPU_HANDLE_ERROR(hipDeviceReset());

	if(checkFlag(argv, argc, "debug")){
	    printf("Finish main loop\n");
	    printf("Time\n");

	    printf("\tredistribute time: %.10f segs.\n", timer[REDISTRIBUTE]);

	    printf("\tpropagate time: %.10f segs.\n", timer[PROPAGATE]);

	    printf("\tbounceback time: %.10f segs.\n", timer[BOUNCEBACK]);

	    printf("\trelaxation time: %.10f segs.\n", timer[RELAXATION]);

	    printf("\texecution time: %.10f segs.\n", timer[EXECUTION]);

	    comp_rey(lattice, time-1, checkFlag(argv, argc, "debug"));
	    printf("End of the execution\n\n");
	}

	if(checkFlag(argv, argc, "check")){
		e_j = check_density(lattice, checkFlag(argv, argc, "debug"));
		e_r =(e_j - e_i) / e_j;

		if(e_r > e_s){
			fprintf(stderr, "Relative error is %.15f > %lg\n", e_r, e_s);
			HANDLE_ERROR();
		}
	}

	if(checkFlag(argv, argc, "out"))
		write_results(flagValueText(argv, argc, "output"), lattice);
	dealloc_lattice(lattice, checkFlag(argv, argc, "debug"));

	free(lattice);

	fprintf(stderr, "%.10f %.10f %.10f %.10f %.10f\n", timer[REDISTRIBUTE], timer[PROPAGATE], timer[BOUNCEBACK], timer[RELAXATION], timer[EXECUTION]);
	return 0;
}
